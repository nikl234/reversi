#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : test0.cu
 Author      : Niklas
 Version     :
 Copyright   : Your copyright notice
 Description : CUDU Reversi possibilities calculator and applicator
 ============================================================================
 */

#include <iostream>
#include <numeric>
#include <stdlib.h>
#include "data.cuh"

static void CheckCudaErrorAux (const char *, unsigned, const char *, hipError_t);
#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)
#define PG_SIZE 64;

/**
 * CUDA kernel that computes the possibilities on the given playgrounds
 */
__global__ void possibilityKernel(short *pg, bool *result, short *player) {
	// calculate the pos on the field
	unsigned idx = threadIdx.x;
	unsigned idy = threadIdx.y;
	unsigned idz = threadIdx.z;
	unsigned id = idx + 8 * idy;

	// how many pgs we have
	unsigned pgPos = blockIdx.x * 64;

	if(pg[pgPos + id] == player[blockIdx.x]) {
		// calculate the dirs
		short dirX = idz / 3 == 2 ? 0 : idz / 3 * 2 - 1;
		short dirY = (idz % 3) == 2 ? 0 : (idz % 3) * 2 - 1;

		// printf("pg(%d) id(%d %d) dir(%d %d)\n", pgPos, idx, idy, dirX, dirY);

		bool found = false;
		for(int i = 1; i < 8; i++) {
			short posX = i * dirX + idx;
			short posY = i * dirY + idy;

			// iterate until the pos is out of the field or
			// until we know that it is a position we care or not care about
			if(posX >= 0 && posX < 8 && posY >= 0 && posY < 8) {
				// look if we care about the current id
				// if we find the other player on the pos its great because we can flip
				if(pg[pgPos + posX + posY * 8] == -player[blockIdx.x]) found = true;
				// if we find it empty and we haven't found an other player yet this dir is useless
				else if(pg[pgPos + posX + posY * 8] == 0 && !found) return;
				//if we find an empty spot and we have seen the other player before thats awesome
				else if(pg[pgPos + posX + posY * 8] == 0 && found) {
					result[pgPos + posX + posY * 8] = true;
					//printf("pg(%d) id(%d %d) dir(%d %d) pos(%d %d)\n", pgPos, idx, idy, dirX, dirY, posX, posY);
					return;
				// if we get to ourself somehow stop searching
				} else if(pg[pgPos + posX + posY * 8] == player[blockIdx.x]) return;
			}
		}
	}
}

/**
 * CUDA kernel that creates the new playground from the touched pos and the old one
 */
__global__ void changeKernel(short2 *poss, short *result, short* player) {
	// get the poss and the dir we are on on the field
	unsigned idx = blockIdx.x;
	unsigned dir = threadIdx.x;

	// how many p we have
	unsigned pgPos = idx * 64;

	// calculate the dirs
	short dirX = dir / 3 == 2 ? 0 : dir / 3 * 2 - 1;
	short dirY = (dir % 3) == 2 ? 0 : (dir % 3) * 2 - 1;

	bool found = false;
	bool dirIsRight = false;

	for(int i = 1; i < 8; i++) {
		short posX = i * dirX + poss[idx].x;
		short posY = i * dirY + poss[idx].y;

		short field = result[pgPos + posX + posY * 8];

		// iterate until the pos is out of the field or
		// until we know that it is a position we care or not care about
		if(posX >= 0 && posX < 8 && posY >= 0 && posY < 8) {
			// look if we care about the current id
			// if we find the other player on the pos its great because we can flip
			if(field == -(player[idx])) {
				found = true;
			}
			// if we find it empty and we haven't found an other player yet this dir is useless
			else if(field == 0) {
				return;
			}
			// if we get to ourself somehow and we found the enemy befor its great
			else if(field == player[idx] && found) {
				dirIsRight = true;
				break;
			} else {
			}
		}
	}

	if(dirIsRight) {
		printf("dirX: %d dirY %d \n", dirX, dirY);

		for(int i = 1; i < 8; i++) {
			short posX = i * dirX + poss[idx].x;
			short posY = i * dirY + poss[idx].y;

			bool end = result[posX + posY * 8] == 0;

			result[posX + posY * 8] = player[idx];

			if(end) return;
		}
	}
}

/**
 * Host function that copies the data and launches the work on GPU
 */
bool *gpuPoss(int size, short *pg, bool *out, short *player)
{
	bool *cpuOut= new bool[64 * size];
	short *gpuPG;
	short *gpuPlayer;
	bool *gpuOut;

	// allocate the mem
	printf("Allocating... \n \n");
	CUDA_CHECK_RETURN(hipMalloc((void **)&gpuPG, sizeof(short) * 64 * size));
	CUDA_CHECK_RETURN(hipMalloc((void **)&gpuOut, sizeof(bool) * 64 * size));
	CUDA_CHECK_RETURN(hipMalloc((void **)&gpuPlayer, sizeof(short) * size));

	// copy the initial values
	CUDA_CHECK_RETURN(hipMemcpy(gpuPG, pg, sizeof(short) * 64 * size, hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMemcpy(gpuOut, out, sizeof(bool) * 64 * size, hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMemcpy(gpuPlayer, player, sizeof(short) * size, hipMemcpyHostToDevice));

	const int blockCount = size;
	const dim3 BLOCK_SIZE(8, 8, 8);
	possibilityKernel<<<blockCount, BLOCK_SIZE>>> (gpuPG, gpuOut, gpuPlayer);

	// Wait for GPU to finish before accessing on host
	hipDeviceSynchronize();

	CUDA_CHECK_RETURN(hipMemcpy(cpuOut, gpuOut, sizeof(bool) * 64 * size, hipMemcpyDeviceToHost));
	CUDA_CHECK_RETURN(hipFree(gpuPG));
	CUDA_CHECK_RETURN(hipFree(gpuOut));
	CUDA_CHECK_RETURN(hipFree(gpuPlayer));
	return cpuOut;
}

/**
 * Host function that copies the data and launches the work on GPU
 */
short *gpuPG(int size, short2 *poss, short *pg, short *player)
{
	short *cpuPG= new short[64 * size];
	short *gpuPG;
	short *gpuPlayer;
	short2 *gpuPoss;

	// allocate the mem
	printf("Allocating... \n \n");
	CUDA_CHECK_RETURN(hipMalloc((void **)&gpuPoss, sizeof(short2) * size));
	CUDA_CHECK_RETURN(hipMalloc((void **)&gpuPG, sizeof(short) * 64 * size));
	CUDA_CHECK_RETURN(hipMalloc((void **)&gpuPlayer, sizeof(short) * size));

	// copy the initial values
	CUDA_CHECK_RETURN(hipMemcpy(gpuPoss, poss, sizeof(short2) * size, hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMemcpy(gpuPG, pg, sizeof(short) * 64 * size, hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMemcpy(gpuPlayer, player, sizeof(short) * size, hipMemcpyHostToDevice));

	const int blockCount = size;
	const int blockSize = 8;
	cout << "Running with " << size << " pgs" << endl;
	changeKernel<<<blockCount, blockSize>>> (gpuPoss, gpuPG, gpuPlayer);

	// Wait for GPU to finish before accessing on host
	hipError_t cudaerr = hipDeviceSynchronize();

	if (cudaerr != hipSuccess) {
		 printf("kernel launch failed with error \"%s\".\n",
			               hipGetErrorString(cudaerr));
	}


	CUDA_CHECK_RETURN(hipMemcpy(cpuPG, gpuPG, sizeof(short) * 64 * size, hipMemcpyDeviceToHost));
	CUDA_CHECK_RETURN(hipFree(gpuPG));
	CUDA_CHECK_RETURN(hipFree(gpuPoss));
	CUDA_CHECK_RETURN(hipFree(gpuPlayer));
	return cpuPG;
}

void initialize_pg(result r, short *data, short* player, int size){
	for(int i = 0; i < size; ++i) {
			row row = r[i];

			player[i] = (row[1].as<short>() % 2) * 2 -1;

			auto arr = row[3].as_array();
			for(int j = 0; j < 64; j++) {
				string s = arr.get_next().second;
				if(s != "") {
					int content = std::stoi(s);

					data[i * 64 + j] = static_cast<short>(content);
				} else {
					j--;
				}
			}
		}
}

void initialize_poss(result r, short *data, short *player, short2 *poss, short* round,int *last_pg, int size) {
	for(int i = 0; i < size; i++) {
		row row = r[i];

		poss[i] = make_short2(row[1].as<short>(), row[2].as<short>());

		player[i] = (row[3].as<short>() % 2) * 2 -1;
		round[i] = row[3].as<short>();
		last_pg[i] = row[0].as<int>();

		// DATA
		auto arr = row[4].as_array();
		for(int j = 0; j < 64; j++) {
			string s = arr.get_next().second;
			if(s != "") {
				int content = std::stoi(s);

				data[i * 64 + j] = static_cast<short>(content);
			} else {
				j--;
			}
		}
	}
}

void calculate_poss(pg pg) {
	// set the max size
	int size = 1;
	result r = pg.get_open_pg(size);

	// if we get less resize it
	size = r.size();

	if(size == 0) {
		cout << "No results for poss" << endl;
		return;
	}

	short *data = new short[64 * size];
	bool *out = (bool*) malloc(64 * size);
	short *player = new short(size);

	initialize_pg(r, data, player, size);

	bool *poss = gpuPoss(size, data, out, player);

	for(int i = 0; i < size; i++) {
		for(int j = 0; j < 64; j++) {
			if(poss[i * 64 + j]) {
				// TODO: add possibility
				int y = j / 8;
				int x = j % 8;
				int id = r[i][0].as<int>();
				pg.insertPoss(id, x, y);

			}
		}
	}

	short gpuSum = std::accumulate (poss, poss + 64 * size, 0);

	/* Verify the results */
	std::cout << "gpuSum = " << gpuSum;

	/* Free memory */
	delete[] data;
	delete[] poss;
}

void calculate_pg(pg pg) {
	int size = 1;

	result r = pg.get_open_poss(size);
	size = r.size();

	if(size == 0) {
		cout << "No results for pg" << endl;
		return;
	}

	short *data = new short[64 * size];
	short *player = new short[size];
	short2 *poss = new short2[size];
	short *round = new short[size];
	int *last_pg = new int[size];

	initialize_poss(r, data, player, poss, round, last_pg, size);

	cout << "Poss(" << poss[0].x << " " << poss[0].y << ")" << endl;
	cout << "Player: " << player[0] << endl;
	short *gpu_pg = gpuPG(size, poss, data, player);

	pg.insertPlayground(gpu_pg, round, last_pg, poss, size);

}

int main(void)
{
	pg pg;
	pg.connect();

	//calculate_poss(pg);
	calculate_pg(pg);

	return 0;
}

/**
 * Check the return value of the CUDA runtime API call and exit
 * the application if the call has failed.
 */
static void CheckCudaErrorAux (const char *file, unsigned line, const char *statement, hipError_t err)
{
	if (err == hipSuccess)
		return;
	std::cerr << statement<<" returned " << hipGetErrorString(err) << "("<<err<< ") at "<<file<<":"<<line << std::endl;
	exit (1);
}

