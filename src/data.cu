#include "hip/hip_runtime.h"
#include "data.cuh"

// todo: add the PostgreSQL implementations here to add and get it out of the DB
result pg::get_open_pg(int size) {
	work w(*c);
	result r = w.exec_prepared("SELECT_PLAYGROUNDS", size);
	return r;
}

result pg::get_open_poss(int size) {
	work w(*c);
	result r = w.exec_prepared("SELECT_POS", size);
	return r;
}

int pg::connect() {
	c = new connection("dbname = reversi user = postgres password = atotallysecurepassword hostaddr = 127.0.0.1 port = 5432");

	if(c->is_open()) {
		cout << "Opened DB: " << (*c).dbname() << endl;

		pg::prepare();

		return 0;
	} else cout << "Can not open DB" << (*c).dbname() << endl;
	return 1;
}

int pg::insertPlayground(short *pg, short *round, int *last_pg, short2 *poss, int size) {
	cout << "Inserting pgs.." << endl;
	work w(*c);

	for(int x = 0; x < size; x++) {
		stringstream s;
		s << "{";

		for(int i = 0; i < 63; i++) {
			s << std::to_string(pg[i]) << ",";
		}
		s << std::to_string(pg[64]) << "}";
		string array = s.str();

		result r = w.exec_prepared("INSERT_PG", round[x] + 1, array);
		w.exec_prepared("UPDATE_POS", r[0][0].as<int>(), last_pg[x], poss[x].x, poss[x].y);
		cout << "New id: " << r[0][0] << " " << array << endl;
	}


	w.commit();

	return 0;
}

int pg::prepare() {
	cout << "Preparing Statements.." << endl;
	cout << c << endl;

	c->prepare("SELECT_PLAYGROUNDS", "SELECT * FROM playground WHERE NOT pos_generated LIMIT $1");
	c->prepare("INSERT_PG", "INSERT INTO playground(round, map) VALUES($1, $2) RETURNING playground");
	c->prepare("UPDATE_PG", "UPDATE playground SET pos_generated = true WHERE playground = $1");
	c->prepare("INSERT_POS", "INSERT INTO link(last_pg, x, y) VALUES($1, $2, $3)");
	c->prepare("UPDATE_POS", "UPDATE link SET next_pg = $1 WHERE last_pg = $2 AND x = $3 AND y = $4");
	c->prepare("SELECT_POS", "SELECT last_pg, x, y, round, map FROM link l LEFT JOIN playground p ON l.last_pg = p.playground WHERE next_pg IS NULL LIMIT $1");

	return 0;
}

int pg::insertPoss(int id, short x, short y) {
	work w(*c);
	w.exec_prepared("INSERT_POS", id, x, y);
	w.exec_prepared("UPDATE_PG", id);
	w.commit();
	return 0;
}
